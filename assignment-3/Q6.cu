#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define SIZE 1024
#define TILE_WIDTH 16

float h_M[SIZE*SIZE],h_N[SIZE*SIZE],h_P[SIZE*SIZE];

__global__ void multiplication_kernel(float *d_M,float *d_N,float *d_P)
{
__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

int tx = threadIdx.x;
int ty = threadIdx.y;
int bx = blockIdx.x;
int by = blockIdx.y;

int col = TILE_WIDTH * bx + tx;
int row = TILE_WIDTH * by + ty;

float prod_value = 0;
int m,k;
for(m=0;m<SIZE/TILE_WIDTH;m++)
{
    ds_M[ty][tx] = d_M[row*SIZE+(m*TILE_WIDTH+tx)];
    ds_N[ty][tx] = d_N[(m*TILE_WIDTH+ty)*SIZE+col];
    __syncthreads();

    for(k=0;k<TILE_WIDTH;k++)
        prod_value+=ds_M[ty][k]*ds_N[k][tx];
    __syncthreads();
}
d_P[row*SIZE+col] = prod_value;
}

void matrix_multiplication(float *d_M,float *d_N,float *d_P)
{
    dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
    dim3 dimGrid(SIZE/TILE_WIDTH,SIZE/TILE_WIDTH,1);

    multiplication_kernel<<<dimGrid,dimBlock>>>(d_M,d_N,d_P);
}

void display_matrix(float mat[])
{
    int i,j;
    for(i=0;i<SIZE;i++)
    {
        for(j=0;j<SIZE;j++)
            printf("%f ",mat[i*SIZE+j]);
        printf("\n");
    }
}

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr,"No devices supporting cuda\n");
        exit(EXIT_FAILURE);
    }
    int deviceId = 0;
    hipSetDevice(deviceId);
    
    const int ARRAY_BYTES = SIZE*SIZE*sizeof(float);
    float *d_M,*d_N,*d_P;
    hipMalloc((void**)&d_M,ARRAY_BYTES);
    hipMalloc((void**)&d_N,ARRAY_BYTES);
    hipMalloc((void**)&d_P,ARRAY_BYTES);

    int i,j;
    for(i=0;i<SIZE;i++)
    {
        for(j=0;j<SIZE;j++)
        {
            h_M[i*SIZE+j] = rand()%101;
            h_N[i*SIZE+j] = rand()%101;
        }
    }
    hipMemcpy(d_M,h_M,ARRAY_BYTES,hipMemcpyHostToDevice);
    hipMemcpy(d_N,h_N,ARRAY_BYTES,hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    matrix_multiplication(d_M,d_N,d_P);
    hipEventRecord(stop,0);
    
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipMemcpy(h_P,d_P,ARRAY_BYTES,hipMemcpyDeviceToHost);
    /*
    printf("M is \n");
    display_matrix(h_M);
    printf("N is \n");
    display_matrix(h_N);
    printf("Product of M and N is \n");
    display_matrix(h_P);
    */
    printf("Elapsed time is %f\n",elapsedTime);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);
    return 0;
}