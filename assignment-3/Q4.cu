#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define CHANNELS 3
#define BLUR_SIZE 3
#define SIZE 12
#define PI 3.14

float h_image[SIZE*SIZE*CHANNELS],h_filter[BLUR_SIZE*BLUR_SIZE],h_blurredimage[SIZE*SIZE*CHANNELS];

__global__ void gaussian_blur(float *d_image,float *d_blurredimage,float* d_filter)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int px = bx*blockDim.x+tx;
    int py = by*blockDim.y+ty;

    float c;
    int fx,fy,ch,imgx,imgy;
    for(ch=0;ch<CHANNELS;ch++)
    {
        c=0;
        for(fx=0;fx<BLUR_SIZE;fx++)
        {
            for(fy=0;fy<BLUR_SIZE;fy++)
            {
                imgx = px + fx - BLUR_SIZE/2;
                imgy = py + fy - BLUR_SIZE/2;
                imgx = min(max(imgx,0),SIZE-1);
                imgy = min(max(imgy,0),SIZE-1);
                c+=d_filter[fy*BLUR_SIZE+fx]*d_image[imgy*SIZE*CHANNELS+imgx*CHANNELS+ch];
            }
        }
        d_blurredimage[py*SIZE*CHANNELS+px*CHANNELS+ch] = c;
    }
}
void blur(float *d_image,float *d_blurredimage,float *d_filter)
{
    dim3 dimBlock(4,4,1);
    dim3 dimGrid(SIZE/dimBlock.x,SIZE/dimBlock.y);
    gaussian_blur<<<dimGrid,dimBlock>>>(d_image,d_blurredimage,d_filter);
}

void initialise_image()
{
    int i,j,k;
    for(i=0;i<SIZE;i++)
    {
        for(j=0;j<SIZE;j++)
        {
            for(k=0;k<CHANNELS;k++)
            h_image[i*SIZE*CHANNELS + j*CHANNELS + k] =rand()%256;
        }
    }
}
void initialise_filter()
{
    int i,j;
    double sum = 0.0,r,s=1/(2*PI);

    for(i=-BLUR_SIZE/2;i<=BLUR_SIZE/2;i++)
    {
        for(j=-BLUR_SIZE/2;j<=BLUR_SIZE/2;j++)
        {
        r = exp(-(i*i+j*j))/2;
        h_filter[(i+2)*BLUR_SIZE+(j+2)] = r*s;
        sum +=h_filter[(i+2)*BLUR_SIZE+(j+2)];
        }  
    }
    for(i=0;i<BLUR_SIZE;i++)
    {
        for(j=0;j<BLUR_SIZE;j++)
            h_filter[i*BLUR_SIZE+j]/=sum;
    }
}
int main(int argc, char *argv[])
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr,"No devices supporting cuda\n");
        exit(EXIT_FAILURE);
    }
    int deviceId = 0;
    hipSetDevice(deviceId);

    printf("Initialising\n");
    initialise_image();
    initialise_filter();

    float *d_image,*d_blurredimage,*d_filter;

    const int ARRAY_BYTES = SIZE*SIZE*CHANNELS*sizeof(float);
    const int FILTER_BYTES = BLUR_SIZE*BLUR_SIZE*sizeof(float);

    hipMalloc((void**)&d_image,ARRAY_BYTES);
    hipMalloc((void**)&d_blurredimage,ARRAY_BYTES);
    hipMalloc((void**)&d_filter,FILTER_BYTES);

    hipMemcpy(d_image,h_image,ARRAY_BYTES,hipMemcpyHostToDevice);
    hipMemcpy(d_filter,h_filter,FILTER_BYTES,hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    blur(d_image,d_blurredimage,d_filter);
    hipEventRecord(stop,0);
    
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipMemcpy(h_blurredimage,d_blurredimage,ARRAY_BYTES,hipMemcpyDeviceToHost);
    printf("Elapsed time is %f\n",elapsedTime);
    
    hipFree(d_image);
    hipFree(d_blurredimage);
    return 0;
}