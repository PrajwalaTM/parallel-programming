#include<stdio.h>

int main()
{
    int num_devices,i;
    
    hipGetDeviceCount(&num_devices);
    for(i=0;i<num_devices;i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop,i);
        printf("Device Number: %d\n",i);
        printf("Device Name: %s\n",prop.name);
        printf("Compute Capability: Major - %d Minor - %d\n",prop.major,prop.minor);
        printf("Memory Clock Rate(kHz): %d\n",prop.memoryClockRate);
        printf("Memory Bus Width:(bits): %d\n",prop.memoryBusWidth);
        printf("Multiprocessor Count: %d\n",prop.multiProcessorCount);
        printf("Total Global Memory Size(MB): %zu\n",prop.totalGlobalMem/(1024*1024));
        printf("Total Shared Memory Size(KB): %zu\n",prop.sharedMemPerBlock/(1024));
        printf("Total Constant Memory Size(KB): %zu\n",prop.totalConstMem/(1024));
        printf("Number of Registers Per Block: %d\n",prop.regsPerBlock);
        printf("Warp Size: %d\n",prop.warpSize);
        printf("Maximum Number of Threads Per Block: %d\n",prop.maxThreadsPerBlock);
        printf("Maximum Size of a Block: %d %d %d\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
        printf("Maximum Size of Grid: %d %d %d\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
        printf("Clock Rate(kHz): %f\n",prop.clockRate);
    }
    return 0;
}