#include<stdio.h>
#include<hip/hip_runtime.h>

#define SIZE 1000
int h_M[SIZE*SIZE],h_N[SIZE*SIZE],h_S[SIZE*SIZE];

__global__ void addition_kernel(int *d_M,int *d_N,int *d_S)
{
    int tIdx = blockDim.x*blockIdx.x + threadIdx.x;
    int tIdy = blockDim.y*blockIdx.y + threadIdx.y;
    int s= d_M[tIdy*SIZE+tIdx] + d_N[tIdy*SIZE+tIdx];
    d_S[tIdy*SIZE+tIdx] = s;
}
void matrix_addition(int *d_M,int *d_N,int *d_S)
{
    dim3 dimBlock(32,32,1);
    dim3 dimGrid(SIZE/dimBlock.x,SIZE/dimBlock.y);
    addition_kernel<<<dimGrid,dimBlock>>>(d_M,d_N,d_S);
}
void display_matrix(int mat[])
{
    int i,j;
    for(i=0;i<SIZE;i++)
    {
        for(j=0;j<SIZE;j++)
            printf("%d ",mat[i*SIZE+j]);
        printf("\n");
    }
}
int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr,"No devices supporting cuda\n");
        exit(EXIT_FAILURE);
    }
    int deviceId = 0;
    hipSetDevice(deviceId);
    
    const int ARRAY_BYTES = SIZE*SIZE*sizeof(int);
    int *d_M,*d_N,*d_S;
    hipMalloc((void**)&d_M,ARRAY_BYTES);
    hipMalloc((void**)&d_N,ARRAY_BYTES);
    hipMalloc((void**)&d_S,ARRAY_BYTES);

    int i,j;
    for(i=0;i<SIZE;i++)
    {
        for(j=0;j<SIZE;j++)
        {
            h_M[i*SIZE+j] = i*SIZE+j;
            h_N[i*SIZE+j] = j*SIZE+i;
        }
    }
    hipMemcpy(d_M,h_M,ARRAY_BYTES,hipMemcpyHostToDevice);
    hipMemcpy(d_N,h_N,ARRAY_BYTES,hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    matrix_addition(d_M,d_N,d_S);
    hipEventRecord(stop,0);
    
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipMemcpy(h_S,d_S,ARRAY_BYTES,hipMemcpyDeviceToHost);
    /*printf("M is \n");
    display_matrix(h_M);
    printf("N is \n");
    display_matrix(h_N);
    printf("Addition of M and N is \n");
    display_matrix(h_S);*/

    printf("Elapsed time is %f\n",elapsedTime);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_S);
    return 0;
}