#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

const unsigned int BIN_CAP = 127;
const unsigned int BIN_COUNT = 4096;
const unsigned int ARRAY_SIZE = 1048576;

__global__ void clean_bins(unsigned int *d_bins)
{
    unsigned int threadId = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int item = d_bins[threadId];
    if(item>BIN_CAP)
        d_bins[threadId] = BIN_CAP;
}

__global__ void local_mem_histo(unsigned int num,unsigned int* d_data,unsigned int* d_bins)
{
    unsigned int threadId = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int local_bins[BIN_COUNT];
    memset(local_bins,0,sizeof(local_bins));
    unsigned int i,item;
    for (i=0;i<num;i++)
    {
        item = d_data[threadId*num+i];
        local_bins[item]++;
    }
    for(i=0;i<BIN_COUNT;i++)
        atomicAdd(&(d_bins[i]),local_bins[i]);
}

void histo(unsigned int* d_data,unsigned int* d_bins)
{
    unsigned int threads = 1024;
    unsigned int blocks = 1;
    unsigned int num = ARRAY_SIZE/threads;
    local_mem_histo<<<blocks,threads>>>(num,d_data,d_bins);

    threads=1024;
    blocks = BIN_COUNT/threads;
    clean_bins<<<blocks,threads>>>(d_bins);
}

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr,"No devices supporting cuda\n");
        exit(EXIT_FAILURE);
    }
    int deviceId = 0;
    hipSetDevice(deviceId);
   
    const unsigned int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned int);
    const unsigned int BIN_BYTES = BIN_COUNT * sizeof(unsigned int);

    unsigned int h_data[ARRAY_SIZE];
    unsigned int i;
    for(i=0;i<ARRAY_SIZE;i++)
        h_data[i] = rand()%BIN_COUNT;
    
    unsigned int h_bins[BIN_COUNT];

    unsigned int *d_data,*d_bins;
    hipMalloc((void**)&d_data,ARRAY_BYTES);
    hipMalloc((void**)&d_bins,BIN_BYTES);

    hipMemcpy(d_data,h_data,ARRAY_BYTES,hipMemcpyHostToDevice);
    hipMemset((void**)&d_bins, 0, BIN_BYTES);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

     //Launch the reduce kernel
     hipEventRecord(start,0);
     histo(d_data,d_bins);
     hipEventRecord(stop,0);
     
     hipEventSynchronize(stop);
     float elapsedTime;
     hipEventElapsedTime(&elapsedTime,start,stop);
 
     hipMemcpy(h_bins,d_bins,BIN_BYTES,hipMemcpyDeviceToHost);
     printf("Value of the bins are\n");
     for(i=0;i<BIN_COUNT;i++)
         printf("%u ",h_bins[i]);
     printf("\nElapsed Time is %f\n",elapsedTime);
 
     hipFree(d_data);
     hipFree(d_bins);
     return 0;
 }