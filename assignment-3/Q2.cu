#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>


__global__ void reduce_kernel(float *d_out,float *d_in)
{
    //Size of shared memory is set by third parameter of kernel launch
    extern __shared__ float shared_array[];

    int globalThreadId = threadIdx.x + blockDim.x*blockIdx.x;
    int blockThreadId = threadIdx.x;

    shared_array[blockThreadId] = d_in[globalThreadId];
    int s;

    //Sync to ensure full shared_array is loaded
    __syncthreads();

    //Actual reduction operation
    for(s = blockDim.x/2;s>0;s/=2)
    {
        if(blockThreadId<s)
            shared_array[blockThreadId]+=shared_array[blockThreadId+s];
        __syncthreads();
    }
    //Output of reduction is written to first index of global memory
    if(blockThreadId==0)
        d_out[blockIdx.x] = shared_array[0];
}
void reduce(float *d_in,float *d_intermediate,float *d_out,int array_size)
{
    int threads = 256;
    int blocks = array_size/threads;
    reduce_kernel<<<blocks,threads,threads*sizeof(float)>>>(d_intermediate,d_in);

    //Results of all blocks are stored in one block, which has to be reduced
    threads = blocks;
    blocks = 1;
    reduce_kernel<<<blocks,threads,threads*sizeof(float)>>>(d_out,d_intermediate);
}
int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr,"No devices supporting cuda\n");
        exit(EXIT_FAILURE);
    }
    int deviceId = 0;
    hipSetDevice(deviceId);
    
    const int ARRAY_SIZE = 1024;
    const int ARRAY_BYTES = ARRAY_SIZE*sizeof(float);

    float h_in[ARRAY_SIZE];
    int i;
    for(i=0;i<ARRAY_SIZE;i++)
        h_in[i]=i;
    
    float *d_in,*d_out,*d_intermediate;

    hipMalloc((void**)&d_in,ARRAY_BYTES);
    hipMalloc((void**)&d_intermediate,ARRAY_BYTES);
    hipMalloc((void**)&d_out,sizeof(float));

    hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Launch the reduce kernel
    hipEventRecord(start,0);
    reduce(d_in,d_intermediate,d_out,ARRAY_SIZE);
    hipEventRecord(stop,0);
    
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    float h_out;
    hipMemcpy(&h_out,d_out,sizeof(float),hipMemcpyDeviceToHost);
    printf("Sum of all array elements is %f\nElapsed time is %f\n",h_out,elapsedTime);
    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);
    return 0;
}