#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define CHANNELS 3
#define SIZE 256
#define PI 3.14

float h_rgb[SIZE*SIZE*CHANNELS],h_grayscale[SIZE*SIZE];

__global__ void grayscale_kernel(float *d_rgb,float *d_grayscale)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int px = bx*blockDim.x+tx;
    int py = by*blockDim.y+ty;

    int ch,i,j;
    float r = d_rgb[py*SIZE*CHANNELS+px*CHANNELS+0];
    float g = d_rgb[py*SIZE*CHANNELS+px*CHANNELS+1];
    float b = d_rgb[py*SIZE*CHANNELS+px*CHANNELS+2];

    d_grayscale[py*SIZE+px] = 0.21*r + 0.71*g + 0.07*b; 
}

void convert_rgb_grayscale(float *d_rgb,float *d_grayscale)
{
    dim3 dimBlock(32,32,1);
    dim3 dimGrid(SIZE/dimBlock.x,SIZE/dimBlock.y);
    grayscale_kernel<<<dimGrid,dimBlock>>>(d_rgb,d_grayscale);
}

void initialise_image()
{
    int i,j,k;
    for(i=0;i<SIZE;i++)
    {
        for(j=0;j<SIZE;j++)
        {
            for(k=0;k<CHANNELS;k++)
            h_rgb[i*SIZE*CHANNELS + j*CHANNELS + k] =rand()%256;
        }
    }
}

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if(!deviceCount){
        fprintf(stderr,"No devices supporting cuda\n");
        exit(EXIT_FAILURE);
    }
    int deviceId = 0;
    hipSetDevice(deviceId);

    initialise_image();
    float *d_rgb,*d_grayscale;

    const int RGB_BYTES = SIZE*SIZE*CHANNELS*sizeof(float);
    const int GRAYSCALE_BYTES = SIZE*SIZE*sizeof(float);

    hipMalloc((void**)&d_rgb,RGB_BYTES);
    hipMalloc((void**)&d_grayscale,GRAYSCALE_BYTES);

    hipMemcpy(d_rgb,h_rgb,RGB_BYTES,hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    convert_rgb_grayscale(d_rgb,d_grayscale);
    hipEventRecord(stop,0);
    
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipMemcpy(h_grayscale,d_grayscale,GRAYSCALE_BYTES,hipMemcpyDeviceToHost);
    printf("Elapsed time is %f\n",elapsedTime);
    
    hipFree(d_grayscale);
    hipFree(d_rgb);
    return 0;
}
